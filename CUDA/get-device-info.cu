#include <stdio.h>
#include <stdlib.h>

int main() {
  hipDeviceProp_t prop;

  int count;
  hipGetDeviceCount( &count );
  printf("there are %d device(s)\n", count);
  for (int i=0; i<count; ++i) {
    hipGetDeviceProperties( &prop, i );
    printf("name is %s\n", prop.name);
    printf("major.minor is %d.%d\n", prop.major, prop.minor);
    printf("multiProcessorCount is %d\n", prop.multiProcessorCount);
    printf("warpSize is %d\n", prop.warpSize);
    printf("maxThreadsPerBlock is %d\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim is (%d, %d, %d)\n", prop.maxThreadsDim[0],
           prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize is (%d, %d, %d)\n", prop.maxGridSize[0],
           prop.maxGridSize[1], prop.maxGridSize[2]);
    if ( prop.deviceOverlap )
      printf("device overlap is enabled\n");
    else
      printf("device overlap is NOT enabled\n");
  }

  return 0;
}
