#include "hip/hip_runtime.h"
//Alex Shapovalov
//CS 3220
//Programming Assignment #5, CUDA Matrix Vector

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

const int BLOCK_SIZE = 16;

//block number = n / threads per block

__global__
void MxV( float *M, float *x, float *y, int n) {
    const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
    if (tidx < n && tidy < n) {
        z[tidx*n + tidy] = x[tidx*n + tidy] + y[tidx*n + tidy];
        //tidx = m[0] * x[0] + m[1] * x[1] + m[2] * x[2]
        //no partial sums / local cache
}

int main() {
    float *d_x, *d_y, *d_z;
    size_t pitch;
    hipMalloc((void**) &d_x, N*N*sizeof(float));
    hipMalloc((void**) &d_y, N*N*sizeof(float));
    hipMalloc((void**) &d_z, N*N*sizeof(float));
    hipMemcpy(d_x, h_x, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N*N*sizeof(float), hipMemcpyHostToDevice);

    dim3 blocks(1, 1, 1);
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    blocks.x = (N + BLOCK_SIZE – 1) / BLOCK_SIZE;
    blocks.y = (N + BLOCK_SIZE – 1) / BLOCK_SIZE;
    add2D<<<blocks, threadsPerBlock>>>( d_x, d_y, d_z, N );
    hipDeviceSynchronize(); // this blocks until the device has completed all requested tasks

    //vector norms
    //compare results?


    return 0;
}
