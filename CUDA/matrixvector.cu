#include "hip/hip_runtime.h"
//Alex Shapovalov
//CS 3220
//Programming Assignment #5, CUDA Matrix Vector

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

const int BLOCK_SIZE = 16;

__global__
void MxV( float *x, float *y, float *z, int n ) {
    const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
    if (tidx < n && tidy < n) {
        int i = tidx * n + tidy;
        z[i] = x[i] + y[i];
    }
}

int main() {
    int N = 5000;

    srand48(time(0));

    float *x = (float*)malloc(N*N*sizeof(float));
    float *y = (float*)malloc(N*N*sizeof(float));
    float *z = (float*)malloc(N*N*sizeof(float));

    //fill matrices
    for (int i = 0; i < N*N; i++) {
        x[i] = drand48();
        y[i] = drand48();
    }

    float *d_x, *d_y, *d_z;
    size_t pitch;
    hipMalloc((void**) &d_x, N*N*sizeof(float));
    hipMalloc((void**) &d_y, N*N*sizeof(float));
    hipMalloc((void**) &d_z, N*N*sizeof(float));



    //time without memory
    //------------------------------------------------------------------------------------------------------------------



    hipMemcpy(d_x, x, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*N*sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    dim3 blocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);

    MxV<<<blocks, threadsPerBlock>>>( d_x, d_y, d_z, N );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTimeGPUNoMem;
    hipEventElapsedTime(&elapsedTimeGPUNoMem, start, stop);
    printf("elapsed time GPU with no memory copies: %.4f ms\n", elapsedTimeGPUNoMem);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceSynchronize();
    hipMemcpy( z, d_z, N*N*sizeof(float), hipMemcpyDeviceToHost );



    //time with memory
    //------------------------------------------------------------------------------------------------------------------

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMemcpy( d_x, x, N*N*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( d_y, y, N*N*sizeof(float), hipMemcpyHostToDevice );

    MxV<<<blocks, threadsPerBlock>>>( d_x, d_y, d_z, N );

    hipDeviceSynchronize();
    hipMemcpy( z, d_z, N*N*sizeof(float), hipMemcpyDeviceToHost );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTimeGPUMem;
    hipEventElapsedTime(&elapsedTimeGPUMem, start, stop);
    printf("elapsed time GPU with memory copies: %.4f ms\n", elapsedTimeGPUMem);
    hipEventDestroy(start);
    hipEventDestroy(stop);



    //------------------------------------------------------------------------------------------------------------------



    // finish up on the CPU side
    struct timeval t1, t2;
    float elapsedTimeCPU;
    gettimeofday(&t1, NULL);

    //cpu multiplication here
    float *cpu_z = (float*)malloc(N*N*sizeof(float));
    for (int i = 0; i < N*N; i++) {
        cpu_z[i] = x[i] + y[i];
    }

    gettimeofday(&t2, NULL);
    elapsedTimeCPU = (t2.tv_sec - t1.tv_sec) * 1000.0; // sec to ms
    elapsedTimeCPU += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
    printf("elapsed time CPU: %f ms\n", elapsedTimeCPU); // elapsed time in milliseconds

    //compare vectors relative error
        float *d = (float *)malloc( N*N*sizeof(float));
        for (int i = 0; i < N*N; i++) {
            d[i] = z[i] - cpu_z[i];
        }

        float sum = 0.0;
        for (int i = 0; i < N*N; i++) {
            sum += d[i] * d[i];
        }

        float norm_d = sqrt(sum);

        sum = 0.0;
        for (int i = 0; i < N*N; i++) {
            sum += cpu_z[i] * cpu_z[i];
        }

        float norm_cpu_z = sqrt(sum);

    float relative_error = norm_d / norm_cpu_z;
    printf("Relative error = %f\n", relative_error);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    free(x);
    free(y);
    free(z);
    free(cpu_z);
    free(d);

    return 0;
}
