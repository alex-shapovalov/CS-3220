#include "hip/hip_runtime.h"
//Alex Shapovalov
//CS 3220
//Programming Assignment #5, CUDA

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

const int BLOCK_SIZE = 16;

__global__
void dotp( float *U, float *V, float *partialSum, int n ) {
    extern __shared__ float localCache[];
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    localCache[threadIdx.x] = U[tidx] * V[tidx];
    __syncthreads();

    int cacheIndex = threadIdx.x;
    int i = blockDim.x / 2;
    while (i > 0) {
        if (cacheIndex < i) {
            localCache[cacheIndex] = localCache[cacheIndex] + localCache[cacheIndex + i];
        }
        __syncthreads();
        i = i / 2;
    }

    if (cacheIndex == 0) {
        partialSum[blockIdx.x] = localCache[cacheIndex];
    }
}

int main() {
    int numBlocks = 256;
    int threadsPerBlock = 256;
    int N = numBlocks * threadsPerBlock; //array size

    srand48(time(0));

    float *U = (float *) malloc(N * sizeof(float));
    float *V = (float *) malloc(N * sizeof(float));
    float *partialSum = (float *) malloc(numBlocks * sizeof(float));

    float *dev_U, *dev_V, *dev_partialSum;
    hipMalloc(&dev_U, N * sizeof(float));
    hipMalloc(&dev_V, N * sizeof(float));
    hipMalloc(&dev_partialSum, numBlocks * sizeof(float));

    //fill arrays
    for (int i=0; i<N; ++i) {
        U[i] = drand48();
        V[i] = drand48();
    }

    hipMemcpy( dev_U, U, N*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_V, V, N*sizeof(float), hipMemcpyHostToDevice );

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    dotp<<<numBlocks, threadsPerBlock, BLOCK_SIZE * sizeof(float)>>>( dev_U, dev_V, dev_partialSum, N );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTimeGPU;
    hipEventElapsedTime(&elapsedTimeGPU, start, stop);
    printf("elapsed time GPU: %.4f ms\n", elapsedTimeGPU);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceSynchronize(); // wait for GPU threads to complete; again, not necessary but good pratice
    hipMemcpy( partialSum, dev_partialSum, numBlocks*sizeof(float), hipMemcpyDeviceToHost );

    // finish up on the CPU side
    float gpuResult = 0.0;
    for (int i=0; i<numBlocks; ++i)
        gpuResult = gpuResult + partialSum[i];

    struct timeval t1, t2;
    float elapsedTimeCPU;
    gettimeofday(&t1, NULL);

    float cpuResult = 0.0;
    for (int i = 0; i < N; ++i) {
        cpuResult += U[i] * V[i];
    }

    gettimeofday(&t2, NULL);
    elapsedTimeCPU = (t2.tv_sec - t1.tv_sec) * 1000.0; // sec to ms
    elapsedTimeCPU += (t2.tv_usec - t1.tv_usec) / 1000.0; // us to ms
    printf("elapsed time CPU: %f ms\n", elapsedTimeCPU); // elapsed time in milliseconds

    hipFree(dev_U);
    hipFree(dev_V);
    hipFree(dev_partialSum);

    free(U);
    free(V);
    free(partialSum);

    return 0;
}
