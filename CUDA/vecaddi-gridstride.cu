// jdh Spring 2023
// example program for CS222


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 1048576 // this is 2^20

__global__
void add( int *X, int *Y, int *Z, int n) {
  int stride = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i = i + stride)
    Z[i] = X[i] + Y[i];
}

//----------------------------------------------------------------

void printDeviceProperties() {
  int count;
  hipDeviceProp_t prop;

  hipGetDeviceCount( &count );
  printf("system has %d device(s)\n", count);
  for (int i=0; i<count; ++i) {
    hipGetDeviceProperties( &prop, i );
    printf("name is %s\n", prop.name);
    printf("multiProcessorCount is %d\n", prop.multiProcessorCount);
    printf("warpSize is %d\n", prop.warpSize);
    printf("maxThreadsPerBlock is %d\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim is (%d, %d, %d)\n", prop.maxThreadsDim[0],
           prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize is (%d, %d, %d)\n", prop.maxGridSize[0],
           prop.maxGridSize[1], prop.maxGridSize[2]);
  }
}

//----------------------------------------------------------------

int main() {
  int *X, *Y, *Z;
  int *dev_X, *dev_Y, *dev_Z;

  printDeviceProperties();

  // allocate memory on the host
  X = (int *) malloc(N*sizeof(int));
  Y = (int *) malloc(N*sizeof(int));
  Z = (int *) malloc(N*sizeof(int));

  // allocate memory on the GPU
  hipMalloc( (void **) &dev_X, N*sizeof(int) );
  hipMalloc( (void **) &dev_Y, N*sizeof(int) );
  hipMalloc( (void **) &dev_Z, N*sizeof(int) );

  // set up the problem on the host
  for (int i=0; i<N; ++i) {
    X[i] = i;
    Y[i] = i*i;
  }

  // copy data to the GPU
  hipMemcpy( dev_X, X, N*sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy( dev_Y, Y, N*sizeof(int), hipMemcpyHostToDevice );

  int threadsPerBlock = 256;
  int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;
  add<<<numBlocks,threadsPerBlock>>>( dev_X, dev_Y, dev_Z, N);

  // not necessary to force explicity synchronization between GPU and host--
  // synchronization will happen by default

  // copy results back to the host
  hipMemcpy( Z, dev_Z, N*sizeof(int), hipMemcpyDeviceToHost );

  int fail = 0;
  for (int i=0; i<N; ++i) {
    if (Z[i] != X[i] + Y[i])
      fail = 1;
  }

  if (fail)
    printf("error!\n");
  else
    printf("success\n");

  hipFree( dev_X );
  hipFree( dev_Y );
  hipFree( dev_Z );

  free(X);
  free(Y);
  free(Z);

  return 0;
}
